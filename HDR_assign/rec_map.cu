
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void recMapKernel(float *ptr, int width, int height, float contrast, float intensity, float colorc, float lighta, float *adaptGlobal)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < width && y < height)
	{
		int idx = 3 * (width * y + x);
		ptr += idx;
		float lum = 0.2125f * ptr[0] + 0.7154f * ptr[1] + 0.0721f * ptr[2];
		for(int channel = 0; channel < 3; channel++) {

			float adaptLocal = colorc * ptr[channel] + (1.0f - colorc) * lum;
			float adapt = lighta * adaptLocal + (1.0f - lighta) * adaptGlobal[channel];
			ptr[channel] = ptr[channel] / (ptr[channel] + powf(intensity * adapt, contrast));
		}
	}
}

extern "C" 
void recMapApply(float *ptr, int width, int height, float contrast, float intensity, float colorc, float lighta, float *adaptGlobal)
{	
	int image_memory = width * height * 3 * sizeof(*ptr);

	float *gpuPtr = NULL;
	hipMalloc((void**) &gpuPtr, image_memory);
	float *adaptGpu = NULL;
	hipMalloc((void**) &adaptGpu, 3*sizeof(float));
	hipMemcpy(gpuPtr, ptr, image_memory, hipMemcpyHostToDevice);
	hipMemcpy(adaptGpu, adaptGlobal, 3*sizeof(float), hipMemcpyHostToDevice);
	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);

	recMapKernel<<<blocks, threads>>>(gpuPtr, width, height, contrast, intensity, colorc, lighta, adaptGpu);

	hipMemcpy(ptr, gpuPtr, image_memory, hipMemcpyDeviceToHost);
	hipFree(gpuPtr);
}
