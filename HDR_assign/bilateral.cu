#include "hip/hip_runtime.h"
#include "tonemap.h"

__global__ void bilateralKernel(float *imgPtr, float *resPtr, int width, int height, float *kerPtr, float lumSigma, int ker_size)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < width && y < height)
	{
		int center = ker_size / 2;
		int idx = width * y + x;
		resPtr += idx;
		float sum = 0;
		float k = 0;
		float Is = imgPtr[y * width + x];
		for(int kery = 0; kery < ker_size; kery++) {

			for(int kerx = 0; kerx < ker_size; kerx++) {

				if(x - center + kerx >= 0 && x - center + kerx < width &&
				   y - center + kery >= 0 && y - center + kery < height)
			    	{
						float Ip = imgPtr[(y - center + kery) * width + x - center + kerx];
						float a = kerPtr[kery * ker_size + kerx] * exp(-(Ip - Is) * (Ip - Is) / (2.0f * lumSigma));
						sum += a * Ip;					
						k += a;
					}
				}
			}
			if(k > 0)
				*resPtr = sum / k;
			else
				*resPtr = 0;
	}
}

extern "C" 
void bilateralFilter(HDRImage *img, HDRImage *res, float lumSigma, float spaceSigma)
{
	int ker_size = 10;//((spaceSigma - 0.8) / 0.3 + 1) * 2;
	int width = img->getWidth();
	int height = img->getHeight();
	HDRImage *kernel = makeGaussian(ker_size, ker_size, spaceSigma);
	
	float *gpuPtr = NULL;
	float *gpuKer = NULL;
	float *gpuRes = NULL;
	hipMalloc((void**) &gpuPtr, width * height * sizeof(float));
	hipMalloc((void**) &gpuRes, width * height * sizeof(float));
	hipMalloc((void**) &gpuKer, ker_size * ker_size * sizeof(float));
		
	hipMemcpy(gpuPtr, img->getPtr(), width * height * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gpuKer, kernel->getPtr(), ker_size * ker_size * sizeof(float), hipMemcpyHostToDevice);

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);

	bilateralKernel<<<blocks, threads>>>(gpuPtr, gpuRes, width, height, gpuKer, lumSigma, ker_size);

	hipMemcpy(res->getPtr(), gpuRes, width * height * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(gpuPtr);
	hipFree(gpuKer);
	hipFree(gpuRes);
	delete kernel;
}