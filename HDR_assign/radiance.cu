_global__ void radKernel(float *imgPtr, int size, float source_size, float source_mult, float tetha, float cost, float sint, float source_real_size, float power,
                         float *source, float *surface, float *observer)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < width && y < height)
	{
		x += -size + 1;

	}
}

extern "C" 
void doRadiance(float *imgPtr, int size, float source_size, float source_mult, float tetha, float cost, float sint, float source_real_size, float power,
                float *source, float *surface, float *observer)
{
	int wavenum = 81;
	float *gpuPtr = NULL;
	float *gpuSource = NULL;
	float *gpuSurface = NULL;
	float *gpuObserver = NULL;
	hipMalloc((void**) &gpuPtr, size * size * sizeof(float) * 3);
	hipMalloc((void**) &gpuSource, wavenum * sizeof(float));
	hipMalloc((void**) &gpuSurface, wavenum sizeof(float));
	hipMalloc((void**) &gpuObserver, wavenum * sizeof(float) * 3);

	hipMemcpy(gpuPtr, imgPtr, size * size * sizeof(float) * 3, hipMemcpyHostToDevice);

	dim3 threads(16, 16);
	dim3 blocks((size + threads.x - 1) / threads.x, (size + threads.y - 1) / threads.y);

	radKernel<<<blocks, threads>>>(gpuPtr, size, source_size, source_mult, tetha, cost, sint, source_real_size, power,
                                   gpuSource, gpuSurface, gpuObserver);

	hipMemcpy(ptr, gpuPtr, size * size * sizeof(float) * 3, hipMemcpyDeviceToHost);
	hipFree(gpuPtr);
	hipFree(gpuSource);
	hipFree(gpuSurface);
	hipFree(gpuObserver);
}
