
#include <hip/hip_runtime.h>
__device__ float safelog(float x) {

	if(x < 4.6e-5f)
		return -10.0f;
	return logf(x);
}

__global__ void lumKernel(float *ptr, float *lum, int width, int height, float *logged)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < width && y < height)
	{
		int idx = width * y + x;
		ptr += 3 * idx;
		lum += idx;
		logged += idx;

		*lum = 0.2125f * ptr[0] + 0.7154f * ptr[1] + 0.0721f * ptr[2];
		*logged = safelog(*lum);
	}
}

extern "C" 
void makeLum(float *ptr, float *lum, int width, int height, float *logged)
{
	int lum_memory = width * height * sizeof(*ptr);
	int image_memory = lum_memory * 3;

	float *gpuPtr = NULL;
	float *lumPtr = NULL;
	float *logLumPtr = NULL;
	hipMalloc((void**) &gpuPtr, image_memory);
	hipMalloc((void**) &lumPtr, lum_memory);
	hipMalloc((void**) &logLumPtr, lum_memory);
		
	hipMemcpy(gpuPtr, ptr, image_memory, hipMemcpyHostToDevice);

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
	lumKernel<<<blocks, threads>>>(gpuPtr, lumPtr, width, height, logLumPtr);

	hipMemcpy(lum, lumPtr, lum_memory, hipMemcpyDeviceToHost);
	if(logged)
		hipMemcpy(logged, logLumPtr, lum_memory, hipMemcpyDeviceToHost);
	
	hipFree(gpuPtr);
	hipFree(lumPtr);
	hipFree(logLumPtr);
}
