
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__device__ float slog(float x) {

	if(x < 4.6e-5f)
		return -10.0f;
	return logf(x);
}

__global__ void logMapKernel(float *ptr, int width, int height, float av, float logAv, float power, float innerMult, float mult)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < width && y < height)
	{
		int idx = 3 * (width * y + x);
		ptr += idx;
		float lum = 0.2125f * ptr[0] + 0.7154f * ptr[1] + 0.0721f * ptr[2];
		float map = (slog(lum + av) - logAv) / slog(2.0f + powf(lum, power) * innerMult) * mult / lum;
		ptr[0] *= map;
		ptr[1] *= map;
		ptr[2] *= map;
	}
}

extern "C" 
void logMapApply(float *ptr, int width, int height, float av, float logAv, float power, float innerMult, float mult)
{	
	int image_memory = width * height * 3 * sizeof(*ptr);

	float *gpuPtr = NULL;
	hipMalloc((void**) &gpuPtr, image_memory);
		
	hipMemcpy(gpuPtr, ptr, image_memory, hipMemcpyHostToDevice);

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);

	logMapKernel<<<blocks, threads>>>(gpuPtr, width, height, av, logAv, power, innerMult, mult);

	hipMemcpy(ptr, gpuPtr, image_memory, hipMemcpyDeviceToHost);
	hipFree(gpuPtr);
}
