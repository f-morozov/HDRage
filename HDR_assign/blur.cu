
#include <hip/hip_runtime.h>

__global__ void gaussKernel(float *ptr, int width, int height, int sigma2)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < width && y < height)
	{
		int xc = width / 2;
		int yc = height / 2;
		int idx = (width * y + x);
		ptr += idx;
		float pi = 3.1415926f;
		*ptr = expf(-((x - xc)*(x - xc)+(y - yc)*(y - yc)) / (2*sigma2)) / (2 * pi * sigma2);
	}
}

extern "C"
float *makeGaussianGpu(int width, int height, float sigma)
{
	float *ptr = NULL;
	hipMalloc((void**) &ptr, width * height * sizeof(float));

	float sigma2 = sigma * sigma;
	
	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);

	gaussKernel<<<blocks, threads>>>(ptr, width, height, sigma2);

	return ptr;
}
