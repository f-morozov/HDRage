
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void channelScaleKernel(float *ptr, int width, int height, float min_val, float scale, float new_min)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < width && y < height)
	{
		int idx = 3 * (width * y + x);
		ptr[idx] = new_min + (ptr[idx] - min_val) * scale;
		ptr[idx + 1] = new_min + (ptr[idx + 1] - min_val) * scale;
		ptr[idx + 2] = new_min + (ptr[idx + 2] - min_val) * scale;
	}
}

extern "C" 
void scaleChannels(float *ptr, int width, int height, float min_val, float scale, float new_min)
{	
	int image_memory = width * height * 3 * sizeof(*ptr);

	float *gpuPtr = NULL;
	hipMalloc((void**) &gpuPtr, image_memory);
		
	hipMemcpy(gpuPtr, ptr, image_memory, hipMemcpyHostToDevice);

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);

	channelScaleKernel<<<blocks, threads>>>(gpuPtr, width, height, min_val, scale, new_min);

	hipMemcpy(ptr, gpuPtr, image_memory, hipMemcpyDeviceToHost);
	hipFree(gpuPtr);
}
