
#include <hip/hip_runtime.h>
__global__ void gammaKernel(float *ptr, int width, int height, float invGamma)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < width && y < height)
	{
		int idx = 3 * (width * y + x);
		ptr[idx] = powf(ptr[idx], invGamma);
		ptr[idx + 1] = powf(ptr[idx + 1], invGamma);
		ptr[idx + 2] = powf(ptr[idx + 2], invGamma);
	}
}

extern "C" 
void applyGamma(float *ptr, int width, int height, float invGamma)
{
	int image_memory = width * height * 3 * sizeof(*ptr);

	float *gpuPtr = NULL;
	hipMalloc((void**) &gpuPtr, image_memory);
		
	hipMemcpy(gpuPtr, ptr, image_memory, hipMemcpyHostToDevice);

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);

	gammaKernel<<<blocks, threads>>>(gpuPtr, width, height, invGamma);

	hipMemcpy(ptr, gpuPtr, image_memory, hipMemcpyDeviceToHost);
	hipFree(gpuPtr);
}